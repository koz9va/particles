#include <iostream>
#include "hip/hip_runtime.h"

#include "hiprand/hiprand.h"
#include <stdio.h>
#include "math.h"

const int am = 10;
const int threadsPerBlock = 256;
const int blocksPerGrid = (am + threadsPerBlock - 1) / threadsPerBlock;
double* particles = new double[am * 5];

// 0 x; 1 y; 2 z; 3 ang0; 4 ang1;

__global__ void process(double* parts, double* result)
{
	int inx = threadIdx.x + blockIdx.x * blockDim.x;
	double sin0;
	result[inx] = parts[inx] + sin(parts[inx+3])*cos(parts[inx+4]);
	
	__syncthreads();

}

int main()
{

	for (int i = 0; i < am; i++) {
		for (int x = 0; x < 5; x++) {
			particles[i + x] = 1;
		}
	}

	double* devparticles;
	double* devres;

	hipMalloc((void**)&devparticles, sizeof(double)*am);
	hipMalloc((void**)&devres, sizeof(double)*am);

	hipMemcpy(devparticles, particles, sizeof(double) * am * 5, hipMemcpyHostToDevice);

	process<<<blocksPerGrid,  threadsPerBlock>>>(devparticles, devres);

	hipMemcpy(particles, devres, sizeof(double) * am * 5, hipMemcpyDeviceToHost);



	hipFree(devparticles);
	return 0;
}
